#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int INF = (1 << 30) - 1;
const int B = 64;
int V, E, nV;
int *dist;

__constant__ int nV_d;

int ceil(int a, int b) {
    return (a + b - 1) / b;
}

void input(char *input_filename) {
    FILE* input_file = fopen(input_filename, "rb");
    fread(&V, sizeof(int), 1, input_file);
    fread(&E, sizeof(int), 1, input_file);

    // printf("V = %d, E = %d\n", V, E);

    nV = ceil(V, B) * B;
    hipHostMalloc((void **)&dist, nV * nV * sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < nV; ++i) {
        for (int j = 0; j < nV; ++j) {
            dist[i * nV + j] = i == j && i < V ? 0 : INF;
        }
    }

    int pair[3];
    for (int i = 0; i < E; ++i) {
        fread(pair, sizeof(int), 3, input_file);
        dist[pair[0] * nV + pair[1]] = pair[2];
    }
    fclose(input_file);
}

void output(char* output_filename) {
    FILE* output_file = fopen(output_filename, "w");
    for (int i = 0; i < V; ++i) {
        fwrite(dist + i * nV, sizeof(int), V, output_file);
    }
    fclose(output_file);
}

//======================
__global__ void phase1(int *dist, int r) {
    int x = threadIdx.y, y = threadIdx.x;
    int i = x + (r << 6), j = y + (r << 6);

    __shared__ int dist_shared[4096];
    dist_shared[(x << 6) + y] = dist[i * nV_d + j];
    dist_shared[(x << 6) + y + 32] = dist[i * nV_d + j + 32];
    dist_shared[((x + 32) << 6) + y] = dist[(i + 32) * nV_d + j];
    dist_shared[((x + 32) << 6) + y + 32] = dist[(i + 32) * nV_d + j + 32];

    #pragma unroll
    for (int k = 0; k < 64; ++k) {
        __syncthreads();
        dist_shared[(x << 6) + y] = min(dist_shared[(x << 6) + y], dist_shared[(x << 6) + k] + dist_shared[(k << 6) + y]);
        dist_shared[(x << 6) + y + 32] = min(dist_shared[(x << 6) + y + 32], dist_shared[(x << 6) + k] + dist_shared[(k << 6) + y + 32]);
        dist_shared[((x + 32) << 6) + y] = min(dist_shared[((x + 32) << 6) + y], dist_shared[((x + 32) << 6) + k] + dist_shared[(k << 6) + y]);
        dist_shared[((x + 32) << 6) + y + 32] = min(dist_shared[((x + 32) << 6) + y + 32], dist_shared[((x + 32) << 6) + k] + dist_shared[(k << 6) + y + 32]);
    }

    dist[i * nV_d + j] = dist_shared[(x << 6) + y];
    dist[i * nV_d + j + 32] = dist_shared[(x << 6) + y + 32];
    dist[(i + 32) * nV_d + j] = dist_shared[((x + 32) << 6) + y];
    dist[(i + 32) * nV_d + j + 32] = dist_shared[((x + 32) << 6) + y + 32];
}

__global__ void phase2(int *dist, int r) {
    int bx = blockIdx.x, by = blockIdx.y >= r ? blockIdx.y + 1 : blockIdx.y;
    int tx = threadIdx.y, ty = threadIdx.x;

    int i = bx ? tx + (by << 6) : tx + (r << 6);
    int j = bx ? ty + (r << 6) : ty + (by << 6);
    int pivot_i = tx + (r << 6);
    int pivot_j = ty + (r << 6);

    __shared__ int dist_shared[8192];
    // First row => current block
    dist_shared[(tx << 6) + ty] = dist[i * nV_d + j];
    dist_shared[(tx << 6) + ty + 32] = dist[i * nV_d + j + 32];
    dist_shared[((tx + 32) << 6) + ty] = dist[(i + 32) * nV_d + j];
    dist_shared[((tx + 32) << 6) + ty + 32] = dist[(i + 32) * nV_d + j + 32];

    int v1 = dist_shared[(tx << 6) + ty];
    int v2 = dist_shared[(tx << 6) + ty + 32];
    int v3 = dist_shared[((tx + 32) << 6) + ty];
    int v4 = dist_shared[((tx + 32) << 6) + ty + 32];
    int offset = 4096;
    
    // Second row => pivot block
    dist_shared[(tx << 6) + ty + offset] = dist[pivot_i * nV_d + pivot_j];
    dist_shared[(tx << 6) + ty + 32 + offset] = dist[pivot_i * nV_d + pivot_j + 32];
    dist_shared[((tx + 32) << 6) + ty + offset] = dist[(pivot_i + 32) * nV_d + pivot_j];
    dist_shared[((tx + 32) << 6) + ty + 32 + offset] = dist[(pivot_i + 32) * nV_d + pivot_j + 32];
    __syncthreads();

    #pragma unroll
    for (int k = 0; k < 64; ++k) {
        v1 = min(v1, dist_shared[(1 - bx) * offset + (tx << 6) + k] + dist_shared[bx * offset + (k << 6) + ty]);
        v2 = min(v2, dist_shared[(1 - bx) * offset + (tx << 6) + k] + dist_shared[bx * offset + (k << 6) + ty + 32]);
        v3 = min(v3, dist_shared[(1 - bx) * offset + ((tx + 32) << 6) + k] + dist_shared[bx * offset + (k << 6) + ty]);
        v4 = min(v4, dist_shared[(1 - bx) * offset + ((tx + 32) << 6) + k] + dist_shared[bx * offset + (k << 6) + ty + 32]);
    }

    dist[i * nV_d + j] = v1;
    dist[i * nV_d + j + 32] = v2;
    dist[(i + 32) * nV_d + j] = v3;
    dist[(i + 32) * nV_d + j + 32] = v4;
}

__global__ void phase3(int *dist, int r) {
    int bx = blockIdx.x >= r ? blockIdx.x + 1 : blockIdx.x;
    int by = blockIdx.y >= r ? blockIdx.y + 1 : blockIdx.y;

    int tx = threadIdx.y, ty = threadIdx.x;
    int i = tx + (bx << 6), j = ty + (r << 6);
    int a = tx + (r << 6), b = ty + (by << 6);
    int c = tx + (bx << 6), d = ty + (by << 6);

    __shared__ int dist_shared[8192];
    // current block
    int v1 = dist[c * nV_d + d];
    int v2 = dist[c * nV_d + d + 32];
    int v3 = dist[(c + 32) * nV_d + d];
    int v4 = dist[(c + 32) * nV_d + d + 32];
    int offset = 4096;

    // First row => pivot column
    dist_shared[(tx << 6) + ty] = dist[i * nV_d + j];
    dist_shared[(tx << 6) + ty + 32] = dist[i * nV_d + j + 32];
    dist_shared[((tx + 32) << 6) + ty] = dist[(i + 32) * nV_d + j];
    dist_shared[((tx + 32) << 6) + ty + 32] = dist[(i + 32) * nV_d + j + 32];

    // Second row => pivot row
    dist_shared[(tx << 6) + ty + offset] = dist[a * nV_d + b];
    dist_shared[(tx << 6) + ty + 32 + offset] = dist[a * nV_d + b + 32];
    dist_shared[((tx + 32) << 6) + ty + offset] = dist[(a + 32) * nV_d + b];
    dist_shared[((tx + 32) << 6) + ty + 32 + offset] = dist[(a + 32) * nV_d + b + 32];
    __syncthreads();

    #pragma unroll
    for (int k = 0; k < 64; ++k) {
        v1 = min(v1, dist_shared[(tx << 6) + k] + dist_shared[(k << 6) + ty + offset]);
        v2 = min(v2, dist_shared[(tx << 6) + k] + dist_shared[(k << 6) + ty + 32 + offset]);
        v3 = min(v3, dist_shared[((tx + 32) << 6) + k] + dist_shared[(k << 6) + ty + offset]);
        v4 = min(v4, dist_shared[((tx + 32) << 6) + k] + dist_shared[(k << 6) + ty + 32 + offset]);
    }

    dist[c * nV_d + d] = v1;
    dist[c * nV_d + d + 32] = v2;
    dist[(c + 32) * nV_d + d] = v3;
    dist[(c + 32) * nV_d + d + 32] = v4;
}

int main(int argc, char *argv[]) {

    input(argv[1]);

    int *dist_d;
    hipMalloc((void **)&dist_d, nV * nV * sizeof(int));
    hipMemcpy(dist_d, dist, nV * nV * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(nV_d), &nV, sizeof(int));

    // also the number of rounds
    const int N = nV / B;

    dim3 grid1(2, N - 1);
    dim3 grid2(N - 1, N - 1);
    dim3 block(32, 32);

    for (int r = 0; r < N; ++r) {
        phase1<<<1, block>>>(dist_d, r);
        phase2<<<grid1, block>>>(dist_d, r);
        phase3<<<grid2, block>>>(dist_d, r);
    }

    hipMemcpy(dist, dist_d, nV * nV * sizeof(int), hipMemcpyDeviceToHost);
    output(argv[2]);

    hipFree(dist_d);
    hipHostFree(dist);

    return 0;
}