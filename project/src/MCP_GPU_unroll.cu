#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define thread_per_block 1024
#define maxN 10000

const long long INF = 2E18;
int N;
long long *dp_host, *cut_host;
int *p_host;

__constant__ int p_constant[maxN + 1];

__host__ __device__ int convertIdx(int len, int i, int N) {
    return len * N + i;
}

void input() {
    FILE *input_file = fopen("testcase", "r");
    fread(&N, sizeof(int), 1, input_file);

    // malloc host memory
    hipHostMalloc((void **) &p_host, (N + 1) * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void **) &dp_host, (N + 1) * N * sizeof(long long), hipHostMallocDefault);
    hipHostMalloc((void **) &cut_host, (N + 1) * N * sizeof(long long), hipHostMallocDefault);

    // read dimension of each matrix to p_host
    fread(p_host, sizeof(int), N + 1, input_file);
    fclose(input_file);

    // initialize dp_host & cut_host
    for (int len = 1; len <= N; ++len) {
        for (int i = 0; i < N; ++i) {
            dp_host[convertIdx(len, i, N)] = len == 1 ? 0 : INF;
            cut_host[convertIdx(len, i, N)] = -1;
        }
    }
}

__global__ void oneThreadPerEntry(long long *dp_device, long long *cut_device, int len, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = i + len - 1;

    if (i >= N || j >= N) {
        return;
    }

    __shared__ long long dp_shared[4 * 1024];

    for (int k = 0; k < 4; ++k) {
        int idx = convertIdx(len - k - 1, i + k + 1, N);
        dp_shared[convertIdx(k, threadIdx.x, blockDim.x)] = 0 <= idx && idx < (N + 1) * N ? dp_device[idx] : INF;
    }

    __syncthreads();

    long long minimum = dp_device[convertIdx(len, i, N)];
    long long cut_point = -1;
    long long i_dim = p_constant[i];
    long long j_dim = p_constant[j + 1];

    int i_offset = blockIdx.x * blockDim.x;

    #pragma unroll
    for (int k = i; k < j; ++k) {
        int left_idx = convertIdx(k - i + 1, i, N);
        int right_idx = convertIdx(j - k, k + 1, N);

        int diff1 = len - k + i - 1;
        int diff2 = len - j + k;
        int bound1 = convertIdx(k - i + 1, i_offset + diff1, N);
        int bound2 = convertIdx(j - k, i_offset + diff2, N);

        long long left_cost = diff1 <= 4 && bound1 <= left_idx ? dp_shared[convertIdx(diff1 - 1, i - diff1 - i_offset, blockDim.x)] : dp_device[left_idx];
        long long right_cost = diff2 <= 4 && bound2 <= right_idx ? dp_shared[convertIdx(diff2 - 1, k + 1 - diff2 - i_offset, blockDim.x)] : dp_device[right_idx];

        long long cost = left_cost + right_cost + 1LL * i_dim * j_dim * p_constant[k + 1];
        cut_point = cost < minimum ? k : cut_point;
        minimum = min(minimum, cost);
    }

    dp_device[convertIdx(len, i, N)] = minimum;
    cut_device[convertIdx(len, i, N)] = cut_point;
}

int main() {

    double execution_time = 0.0;
    auto start = std::chrono::steady_clock::now();

    hipSetDevice(0);
    input();

    long long *dp_device, *cut_device;
    // int *p_device;
    // cudaMalloc((void **) &p_device, (N + 1) * sizeof(int));
    hipMalloc((void **) &dp_device, (N + 1) * N * sizeof(long long));
    hipMalloc((void **) &cut_device, (N + 1) * N * sizeof(long long));

    // cudaMemcpy(p_device, p_host, (N + 1) * sizeof(int), cudaMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(p_constant), p_host, (N + 1) * sizeof(int));
    hipMemcpy(dp_device, dp_host, (N + 1) * N * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(cut_device, cut_host, (N + 1) * N * sizeof(long long), hipMemcpyHostToDevice);

    // TODO: implement the parallel version of matrix chain multiplication here
    // Can try OneThreadPerEntry, OneBlockPerEntry, MultipleBlocksPerEntry
    // Can try shared memory, coalesced memory access, etc.
    // Observe the memory access pattern of each entry (thread) to explore potential optimization

    // OneThreadPerEntry
    for (int len = 2; len <= N; ++len) {
        int num_blocks = (N - len + thread_per_block) / thread_per_block;
        oneThreadPerEntry<<<num_blocks, thread_per_block>>>(dp_device, cut_device, len, N);
    }

    hipMemcpy(dp_host, dp_device, (N + 1) * N * sizeof(long long), hipMemcpyDeviceToHost);
    hipMemcpy(cut_host, cut_device, (N + 1) * N * sizeof(long long), hipMemcpyDeviceToHost);

    auto end = std::chrono::steady_clock::now();
    execution_time += std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

    std::cout << "Execution time: " << execution_time << " ms" << std::endl;
    std::cout << "Minimum number of multiplications: " << dp_host[convertIdx(N, 0, N)] << std::endl;

    // free the memory spaces that are allocated in host and device
    // cudaFree(p_device);
    hipFree(dp_device);
    hipFree(cut_device);

    hipHostFree(p_host);
    hipHostFree(dp_host);
    hipHostFree(cut_host);
}