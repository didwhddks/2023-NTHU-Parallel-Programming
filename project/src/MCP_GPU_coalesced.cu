#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define thread_per_block 256
#define i64 long long

const i64 inf = 2E18;
int N;
i64 *dp_host, *cut_host;
int *p_host;

__host__ __device__ int convertIdx(int i, int j, int N) {
    return i * N + j;
}

__host__ int ceil(int a, int b) {
    return (a + b - 1) / b;
}

__host__ void input() {
    FILE *input_file = fopen("testcase", "r");
    if (fread(&N, sizeof(int), 1, input_file) != 1) {
        std::cerr << "Error reading from file" << std::endl;
        exit(1);
    }

    std::cout << "Number of matrices: " << N << std::endl;

    // allocate host memory for p_host, dp_host, cut_host
    hipHostMalloc((void **) &p_host, (N + 1) * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void **) &dp_host, N * N * sizeof(i64), hipHostMallocDefault);
    hipHostMalloc((void **) &cut_host, N * N * sizeof(i64), hipHostMallocDefault);

    // read dimensions of matrices
    if (fread(p_host, sizeof(int), N + 1, input_file) != N + 1) {
        std::cerr << "Error reading from file" << std::endl;
        exit(1);
    }
    fclose(input_file);

    // for (int i = 0; i <= N; ++i) {
    //     std::cout << "p[" << i << "] = " << p_host[i] << std::endl;
    // }

    // initialize dp_host and cut_host
    for (int i = 0; i < N; ++i) {
        for (int j = i; j < N; ++j) {
            int idx = convertIdx(j - i, i, N);
            dp_host[idx] = i == j ? 0 : inf;
            cut_host[idx] = -1;
        }
    }
}

__global__ void oneThreadPerEntry(i64 *dp_device, i64 *cut_device, int *p_device, int len, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = i + len;

    if (i < N && j < N) {
        for (int k = i; k < j; ++k) {
            i64 cost = dp_device[convertIdx(k - i, i, N)] + dp_device[convertIdx(j - k - 1, k + 1, N)] +
                            1LL * p_device[i] * p_device[k + 1] * p_device[j + 1];
            if (cost < dp_device[convertIdx(len, i, N)]) {
                dp_device[convertIdx(len, i, N)] = cost;
                cut_device[convertIdx(len, i, N)] = k;
            }
        }
    }
}

int main() {
    hipSetDevice(0);
    input();

    i64 *dp_device, *cut_device;
    int *p_device;
    hipMalloc((void **) &p_device, (N + 1) * sizeof(int));
    hipMalloc((void **) &dp_device, N * N * sizeof(i64));
    hipMalloc((void **) &cut_device, N * N * sizeof(i64));

    hipMemcpy(p_device, p_host, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dp_device, dp_host, N * N * sizeof(i64), hipMemcpyHostToDevice);
    hipMemcpy(cut_device, cut_host, N * N * sizeof(i64), hipMemcpyHostToDevice);

    // CUDA event timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start, 0);

    // Launch kernel for each length
    for (int len = 1; len < N; ++len) {
        int num_blocks = ceil(N - len, thread_per_block);
        // std::cout << "Launching kernel for length " << len << " with " << num_blocks << " blocks" << std::endl;
        oneThreadPerEntry<<<num_blocks, thread_per_block>>>(dp_device, cut_device, p_device, len, N);
        // Check if the kernel launch was successful
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        }
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
        }
    }

    // Record stop time
    hipEventRecord(stop, 0);

    // Wait for all GPU tasks to finish
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);

    // Copy results back to host
    hipMemcpy(dp_host, dp_device, N * N * sizeof(i64), hipMemcpyDeviceToHost);
    hipMemcpy(cut_host, cut_device, N * N * sizeof(i64), hipMemcpyDeviceToHost);
    
    // for (int i = 0; i < N; ++i) {
    //     for (int j = i; j < N; ++j) {
    //         int idx = convertIdx(i, j, N);
    //         std::cout << "dp[" << i << "][" << j << "] = " << dp_host[idx] << ", cut = " << cut_host[idx] << std::endl;
    //     }
    // }

    // Print results
    std::cout << "Execution time: " << elapsed_time << " ms" << std::endl;
    std::cout << "Minimum number of multiplications: " << dp_host[convertIdx(N - 1, 0, N)] << std::endl;

    // Cleanup
    hipFree(p_device);
    hipFree(dp_device);
    hipFree(cut_device);

    hipHostFree(p_host);
    hipHostFree(dp_host);
    hipHostFree(cut_host);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
